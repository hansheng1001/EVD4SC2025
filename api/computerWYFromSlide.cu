
#include "computerWYFromSlide.h"


void computerWYFromSlide(hipblasHandle_t cublas_handle, long M, long N, long slideWidth, double *dW, long ldW,
                         double *dY, long ldY, double *work)
{
    long b = slideWidth;
    double done = 1.0;
    double dzero = 0.0;
    double dnegone = -1.0;

    long ldWork = M;

    for (long i = 2 * b; i <= N; i += b)
    {

        hipblasDgemm(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, i - b, b, M,
                    &done, dY, ldY, dW + (i - b) * ldW, ldW, &dzero, work, ldWork);


        hipblasDgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, b, i - b,
                    &dnegone, dW, ldW, work, ldWork, &done, dW + (i - b) * ldW, ldW);
    }
}